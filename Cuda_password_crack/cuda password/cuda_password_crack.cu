#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>

/****************************************************************************
  This program gives an example of a poor way to implement a password cracker
  in CUDA C. It is poor because it acheives this with just one thread, which
  is obviously not good given the scale of parallelism available to CUDA
  programs.
  
  The intentions of this program are:
    1) Demonstrate the use of __device__ and __global__ functions
    2) Enable a simulation of password cracking in the absence of library 
       with equivalent functionality to libcrypt. The password to be found
       is hardcoded into a function called is_a_match.   

  Compile and run with:
    nvcc -o cuda_crack cuda_password_crack.cu
    ./cuda_crack
   
  Dr Kevan Buckley, University of Wolverhampton, 2018
*****************************************************************************/

/****************************************************************************
  This function returns 1 if the attempt at cracking the password is 
  identical to the plain text password string stored in the program. 
  Otherwise,it returns 0.
*****************************************************************************/

__device__ int is_a_match(char *attempt) {
  char plain_password[] = "DS2018";
  
  char *a = attempt;
  char *p = plain_password;
  
  while(*a == *p) {
    if(*a == '\0') {
      return 1;
    }
    a++;
    p++;
  }
  return 0;
}

/****************************************************************************
  The kernel function assume that there will be only one thread and uses 
  nested loops to generate all possible passwords and test whether they match
  the hidden password.
*****************************************************************************/

__global__ void  kernel() {
  char i, j;
  int k, l, m, n;
  
  char password[8];
  password[6] = '\0'; 
  i = blockIdx.x+65;
  j = threadIdx.x+65;
  
  password[0] =i;
  password[1] =j;
  
  
  for(k=48; k<=57; k++){
    for(l=48; l<=57; l++){
     for(m=48; m<=57; m++){
      for(n=48; n<=57; n++){
	password[3] = l;
        password[2] = k;
        password[4] = m;
        password[5] = n;

        if(is_a_match(password)) {
        printf("password found: %s\n", password);
      } else {
        		  
      }
}
}
}
}

}
int time_difference(struct timespec *start, struct timespec *finish, 
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec; 
  long long int dn =  finish->tv_nsec - start->tv_nsec; 

  if(dn < 0 ) {
    ds--;
    dn += 1000000000; 
  } 
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}



int main(){
struct timespec start, finish;   
  long long int time_elapsed;
  clock_gettime(CLOCK_MONOTONIC, &start);

  kernel <<<26, 26>>>();
  hipDeviceSynchronize();

clock_gettime(CLOCK_MONOTONIC, &finish);
time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, 
         (time_elapsed/1.0e9)); 
 
  return 0;
}


