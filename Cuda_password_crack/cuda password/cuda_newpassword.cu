#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/****************************************************************************
  This program gives an example of a poor way to implement a password cracker
  in CUDA C. It is poor because it acheives this with just one thread, which
  is obviously not good given the scale of parallelism available to CUDA
  programs.
  
  The intentions of this program are:
    1) Demonstrate the use of __device__ and __global__ functions
    2) Enable a simulation of password cracking in the absence of library 
       with equivalent functionality to libcrypt. The password to be found
       is hardcoded into a function called is_a_match.   

  Compile and run with:
    nvcc -o cuda_newpassword.cu cuda_newpassword
    ./cuda_crack
   
  Dr Kevan Buckley, University of Wolverhampton, 2018
*****************************************************************************/

/****************************************************************************
  This function returns 1 if the attempt at cracking the password is 
  identical to the plain text password string stored in the program. 
  Otherwise,it returns 0.
*****************************************************************************/

__device__ int is_a_match(char *attempt) {
  char plain_password1[] = "AB1111";
  char plain_password2[] = "CD1222";
  char plain_password3[] = "EF1333";
  char plain_password4[] = "GH1444";


  char *a = attempt;
  char *b = attempt;
  char *c = attempt;
  char *d = attempt;
  char *p1 = plain_password1;
  char *p2 = plain_password2;
  char *p3 = plain_password3;
  char *p4 = plain_password4;

  while(*a == *p1) { 
   if(*a == '\0') 
    {
	printf("Found password: %s\n",plain_password1);
      break;
    }

    a++;
    p1++;
  }
	
  while(*b == *p2) { 
   if(*b == '\0') 
    {
	printf("Found password: %s\n",plain_password2);
      break;
    }

    b++;
    p2++;
  }

  while(*c == *p3) { 
   if(*c == '\0') 
    {
	printf("Found password: %s\n",plain_password3);
      break;
    }

    c++;
    p3++;
  }

  while(*d == *p4) { 
   if(*d == '\0') 
    {
	printf("Found password: %s\n",plain_password4);
      return 1;
    }

    d++;
    p4++;
  }
  return 0;

}


/****************************************************************************
  The kernel function assume that there will be only one thread and uses 
  nested loops to generate all possible passwords and test whether they match
  the hidden password.
*****************************************************************************/

__global__ void  kernel() {
char k,l,m,n;
  
  char password[7];
  password[6] = '\0';

int i = blockIdx.x+65;
int j = threadIdx.x+65;
char firstValue = i; 
char secondValue = j; 
    
password[0] = firstValue;
password[1] = secondValue;
	for(k='0'; k<='9'; k++){
	  for(l='0'; l<='9'; l++){
	   for(m='0'; m<='9'; m++){
	     for(n='0'; n<='9'; n++){
	        password[2] = k;
	        password[3] = l;
	        password[4] = m;
	        password[5] = n; 
	      if(is_a_match(password)) {
		//printf("Success");
	      } 
             else {
	     //printf("tried: %s\n", password);		  
	         }
	      }
	   }
	}
    }
}

int time_difference(struct timespec *start, 
                    struct timespec *finish, 
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec; 
  long long int dn =  finish->tv_nsec - start->tv_nsec; 

  if(dn < 0 ) {
    ds--;
    dn += 1000000000; 
  } 
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}


int main() {

  struct  timespec start, finish;
  long long int time_elapsed;
  clock_gettime(CLOCK_MONOTONIC, &start);

kernel <<<26,26>>>();
  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9)); 

  return 0;
}


